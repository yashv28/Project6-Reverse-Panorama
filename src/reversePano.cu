#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "reversePano.h"
#include "hip/hip_runtime.h"

#define TIMER 1

#define BLOCK_SIZE 512

#ifndef M_PI
#define M_PI       3.14159265358979323846  /* pi */
#endif

#ifndef M_PI_2
#define M_PI_2     1.57079632679489661923  /* pi/2 */
#endif

#define _max(x, y) ((x < y)?y:x)

#define _min(x, y) ((x < y)?x:y)

#define _clip(x, x_min, x_max) (_max(x_min, _min(x, x_max)))

#define _checkCudaStatus(...) {\
	if (cudaStatus != hipSuccess)\
	{\
		fprintf(stderr, __VA_ARGS__);\
		goto Error;\
	}\
}

namespace reversePano {

	IplImage* readImage(char *imageName) {
		IplImage* targetMat = cvLoadImage(imageName, CV_LOAD_IMAGE_COLOR);

		if(targetMat == NULL)
			fprintf(stderr, "cvLoadImage failed!");

		return targetMat;
	}

	int writeImage(char *imageName, IplImage *image) {
		int ret = cvSaveImage(imageName, image);

		if(!ret)
			fprintf(stderr, "cvSaveImage failed!");

		return !ret;
	}


	void pixelWarpKernelCPU(
		const unsigned char *InImageArray,
		unsigned char *OutImageArray,
		const int FaceOffset[6][2],
		const int Face3DCoordsFactor[6][3][2],
		const int Face3DCoordsConstant[6][3],
		const int outBlockWidth,
		int face, int inFaceX, int inFaceY) {

		int inImageWidth = outBlockWidth << 2;
		int inImageHeight = outBlockWidth << 1;
		int outImageWidth = outBlockWidth << 2;
		int outImageHeight = outBlockWidth * 3;

		float inFaceX_2 = (2.f * inFaceX) / outBlockWidth;
		float inFaceY_2 = (2.f * inFaceY) / outBlockWidth;

		int outX = inFaceX + FaceOffset[face][0];
		int outY = inFaceY + FaceOffset[face][1];

		float cubeX = Face3DCoordsConstant[face][0] +
			Face3DCoordsFactor[face][0][0] * inFaceX_2 +
			Face3DCoordsFactor[face][0][1] * inFaceY_2;
		float cubeY = Face3DCoordsConstant[face][1] +
			Face3DCoordsFactor[face][1][0] * inFaceX_2 +
			Face3DCoordsFactor[face][1][1] * inFaceY_2;
		float cubeZ = Face3DCoordsConstant[face][2] +
			Face3DCoordsFactor[face][2][0] * inFaceX_2 +
			Face3DCoordsFactor[face][2][1] * inFaceY_2;

		float theta = atan2(cubeY, cubeX);
		float r = hypot(cubeX, cubeY);
		float phi = atan2(cubeZ, r);

		float uf = (2.f * outBlockWidth * (theta + M_PI) / M_PI);
		float vf = (2.f * outBlockWidth * (M_PI_2 - phi) / M_PI);

		int ui = floor(uf);
		int vi = floor(vf);

		int ui_2 = ui + 1;
		int vi_2 = vi + 1;

		float mu = uf - ui;
		float nu = vf - vi;

	#define cornerA_x (_clip(vi, 0, inImageHeight - 1))
	#define cornerA_y (ui % inImageWidth)

	#define cornerB_x (_clip(vi, 0, inImageHeight - 1))
	#define cornerB_y (ui_2 % inImageWidth)

	#define cornerC_x (_clip(vi_2, 0, inImageHeight - 1))
	#define cornerC_y (ui % inImageWidth)

	#define cornerD_x (_clip(vi_2, 0, inImageHeight - 1))
	#define cornerD_y (ui_2 % inImageWidth)

	#define _getColor(x, y, offset) \
		(InImageArray[(x * inImageWidth + y) * 3 + offset])

		float colorB =
			_getColor(cornerA_x, cornerA_y, 0) * (1 - mu) * (1 - nu) +
			_getColor(cornerB_x, cornerB_y, 0) * mu * (1 - nu) +
			_getColor(cornerC_x, cornerC_y, 0) * (1 - mu) * nu +
			_getColor(cornerD_x, cornerD_y, 0) * mu * nu;

		float colorG =
			_getColor(cornerA_x, cornerA_y, 1) * (1 - mu) * (1 - nu) +
			_getColor(cornerB_x, cornerB_y, 1) * mu * (1 - nu) +
			_getColor(cornerC_x, cornerC_y, 1) * (1 - mu) * nu +
			_getColor(cornerD_x, cornerD_y, 1) * mu * nu;

		float colorR =
			_getColor(cornerA_x, cornerA_y, 2) * (1 - mu) * (1 - nu) +
			_getColor(cornerB_x, cornerB_y, 2) * mu * (1 - nu) +
			_getColor(cornerC_x, cornerC_y, 2) * (1 - mu) * nu +
			_getColor(cornerD_x, cornerD_y, 2) * mu * nu;

	#define _setColor(x, y, offset) \
		OutImageArray[(x * outImageWidth + y) * 3 + offset]

		_setColor(outX, outY, 0) = (int)round(colorB);
		_setColor(outX, outY, 1) = (int)round(colorG);
		_setColor(outX, outY, 2) = (int)round(colorR);
	}


	int convertWithCPU(
		int outBlockWidth,
		unsigned char *inImageArray,
		unsigned char *outImageArray) {

		int faceOffset[6][2] = {
			{0, outBlockWidth},
			{outBlockWidth, 0},
			{outBlockWidth, outBlockWidth},
			{outBlockWidth, outBlockWidth << 1},
			{outBlockWidth, outBlockWidth * 3},
			{outBlockWidth << 1, outBlockWidth}
		};


		int face3DCoordsFactor[6][3][2] = {
			{	// 0 top
				{1, 0},
				{0, 1},
				{0, 0},
			},
			{	// 1 left
				{0, 1},
				{0, 0},
				{-1, 0},
			},
			{	// 2 front
				{0, 0},
				{0, 1},
				{-1, 0},
			},
			{	// 3 right
				{0, -1},
				{0, 0},
				{-1, 0},
			},
			{	// 4 back
				{0, 0},
				{0, -1},
				{-1, 0},
			},
			{	// 5 down
				{-1, 0},
				{0, 1},
				{0, 0},
			},
		};

		int face3DCoordsConstant[6][3] = {
			{-1, -1, 1},// 0 top
			{-1, -1, 1},// 1 left
			{1, -1, 1},	// 2 front
			{1, 1, 1},	// 3 right
			{-1, 1, 1},	// 4 back
			{1, -1, -1}	// 5 down
		};

#if TIMER
		CpuTimer timer;
		timer.Start();
#endif

		for (int face = 0; face < 6; face++)
			for (int i = 0; i < outBlockWidth; i++)
				for (int j = 0; j < outBlockWidth; j++)
					pixelWarpKernelCPU(
						inImageArray,
						outImageArray,
						(int (*) [2])faceOffset,
						(int (*) [3][2])face3DCoordsFactor,
						(int (*) [3])face3DCoordsConstant,
						outBlockWidth,
						face, i, j);
#if TIMER
		timer.Stop();
		printf("CPU Warp Function: %f s.\n", timer.Elapsed());
#endif

		return 0;
	}

	int CPUmain(IplImage *inImageMat, IplImage **outImageMat) {
		int inImageWidth = inImageMat -> width;
		int inImageHeight = inImageMat -> height;

		if (inImageWidth != (inImageHeight << 1)) {
			fprintf(stderr, "This image is not a standard sphere panorama image (2:1) !");
			return 1;
		}

		int outBlockWidth = inImageWidth >> 2;

		int outImageWidth = outBlockWidth * 4;
		int outImageHeight = outBlockWidth * 3;

		unsigned char *inImageArray = (unsigned char *)(inImageMat->imageData);
		unsigned char *outImageArray = (unsigned char *)malloc(sizeof(unsigned char) * outImageWidth * outImageHeight * 3);

		memset(outImageArray, 0, outImageWidth * outImageHeight);

		int ret = convertWithCPU(outBlockWidth, inImageArray, outImageArray);

		if (ret != 0)
		{
			fprintf(stderr, "convertWithCPU failed!");
			return 1;
		}

		*outImageMat = cvCreateImageHeader(cvSize(outImageWidth, outImageHeight), IPL_DEPTH_8U, 3);
		cvSetData(*outImageMat, outImageArray, outImageWidth * 3);

		return 0;
	}

	hipError_t initCuda() {
		hipError_t cudaStatus;

		int num = 0;
		hipDeviceProp_t prop;
		cudaStatus = hipGetDeviceCount(&num);
		for(int i = 0;i<num;i++)
		{
			hipGetDeviceProperties(&prop,i);
		}
		return cudaStatus;
	}

	__global__ void pixelWarpKernelGPU(
		const unsigned char *cudaInImageArray,
		unsigned char *cudaOutImageArray,
		const int cudaFaceOffset[6][2],
		const int cudaFace3DCoordsFactor[6][3][2],
		const int cudaFace3DCoordsConstant[6][3],
		const int outBlockWidth) {

		int inImageWidth = outBlockWidth << 2;
		int inImageHeight = outBlockWidth << 1;
		int outImageWidth = outBlockWidth << 2;
		int outImageHeight = outBlockWidth * 3;
		int face = blockIdx.x;

		int id = blockIdx.y * BLOCK_SIZE + threadIdx.x;

		int inFaceX = id / outBlockWidth;
		int inFaceY = id % outBlockWidth;

		float inFaceX_2 = (2.f * inFaceX) / outBlockWidth;
		float inFaceY_2 = (2.f * inFaceY) / outBlockWidth;

		int outX = inFaceX + cudaFaceOffset[face][0];
		int outY = inFaceY + cudaFaceOffset[face][1];

		float cubeX = cudaFace3DCoordsConstant[face][0] +
			cudaFace3DCoordsFactor[face][0][0] * inFaceX_2 +
			cudaFace3DCoordsFactor[face][0][1] * inFaceY_2;
		float cubeY = cudaFace3DCoordsConstant[face][1] +
			cudaFace3DCoordsFactor[face][1][0] * inFaceX_2 +
			cudaFace3DCoordsFactor[face][1][1] * inFaceY_2;
		float cubeZ = cudaFace3DCoordsConstant[face][2] +
			cudaFace3DCoordsFactor[face][2][0] * inFaceX_2 +
			cudaFace3DCoordsFactor[face][2][1] * inFaceY_2;

		float theta = atan2(cubeY, cubeX);
		float r = hypot(cubeX, cubeY);
		float phi = atan2(cubeZ, r);

		float uf = (2.f * outBlockWidth * (theta + M_PI) / M_PI);
		float vf = (2.f * outBlockWidth * (M_PI_2 - phi) / M_PI);

		int ui = floor(uf);
		int vi = floor(vf);

		int ui_2 = ui + 1;
		int vi_2 = vi + 1;

		float mu = uf - ui;
		float nu = vf - vi;

#define cornerA_x (_clip(vi, 0, inImageHeight - 1))
#define cornerA_y (ui % inImageWidth)

#define cornerB_x (_clip(vi, 0, inImageHeight - 1))
#define cornerB_y (ui_2 % inImageWidth)

#define cornerC_x (_clip(vi_2, 0, inImageHeight - 1))
#define cornerC_y (ui % inImageWidth)

#define cornerD_x (_clip(vi_2, 0, inImageHeight - 1))
#define cornerD_y (ui_2 % inImageWidth)

#define _getColor(x, y, offset) \
		(cudaInImageArray[(x * inImageWidth + y) * 3 + offset])

		float colorB =
			_getColor(cornerA_x, cornerA_y, 0) * (1 - mu) * (1 - nu) +
			_getColor(cornerB_x, cornerB_y, 0) * mu * (1 - nu) +
			_getColor(cornerC_x, cornerC_y, 0) * (1 - mu) * nu +
			_getColor(cornerD_x, cornerD_y, 0) * mu * nu;

		float colorG =
			_getColor(cornerA_x, cornerA_y, 1) * (1 - mu) * (1 - nu) +
			_getColor(cornerB_x, cornerB_y, 1) * mu * (1 - nu) +
			_getColor(cornerC_x, cornerC_y, 1) * (1 - mu) * nu +
			_getColor(cornerD_x, cornerD_y, 1) * mu * nu;

		float colorR =
			_getColor(cornerA_x, cornerA_y, 2) * (1 - mu) * (1 - nu) +
			_getColor(cornerB_x, cornerB_y, 2) * mu * (1 - nu) +
			_getColor(cornerC_x, cornerC_y, 2) * (1 - mu) * nu +
			_getColor(cornerD_x, cornerD_y, 2) * mu * nu;

#define _setColor(x, y, offset) \
		cudaOutImageArray[(x * outImageWidth + y) * 3 + offset]

		_setColor(outX, outY, 0) = (int)round(colorB);
		_setColor(outX, outY, 1) = (int)round(colorG);
		_setColor(outX, outY, 2) = (int)round(colorR);
	}


	hipError_t convertWithCuda(
		int outBlockWidth,
		unsigned char *inImageArray,
		unsigned char *outImageArray) {

		hipError_t cudaStatus;

		int inImageWidth = outBlockWidth << 2;
		int inImageHeight = outBlockWidth << 1;
		int outImageWidth = outBlockWidth << 2;
		int outImageHeight = outBlockWidth * 3;

		int faceOffset[6][2] = {
			{0, outBlockWidth},
			{outBlockWidth, 0},
			{outBlockWidth, outBlockWidth},
			{outBlockWidth, outBlockWidth << 1},
			{outBlockWidth, outBlockWidth * 3},
			{outBlockWidth << 1, outBlockWidth}
		};

		int face3DCoordsFactor[6][3][2] = {
			{	// 0 top
				{1, 0},
				{0, 1},
				{0, 0},
			},
			{	// 1 left
				{0, 1},
				{0, 0},
				{-1, 0},
			},
			{	// 2 front
				{0, 0},
				{0, 1},
				{-1, 0},
			},
			{	// 3 right
				{0, -1},
				{0, 0},
				{-1, 0},
			},
			{	// 4 back
				{0, 0},
				{0, -1},
				{-1, 0},
			},
			{	// 5 down
				{-1, 0},
				{0, 1},
				{0, 0},
			},
		};

		int face3DCoordsConstant[6][3] = {
			{-1, -1, 1},// 0 top
			{-1, -1, 1},// 1 left
			{1, -1, 1},	// 2 front
			{1, 1, 1},	// 3 right
			{-1, 1, 1},	// 4 back
			{1, -1, -1}	// 5 down
		};

		unsigned char *cudaInImageArray;
		unsigned char *cudaOutImageArray;
		int *cudaFaceOffset;
		int *cudaFace3DCoordsFactor;
		int *cudaFace3DCoordsConstant;

		dim3 dimGrid(6, outBlockWidth * outBlockWidth / BLOCK_SIZE);
		dim3 dimBlock(BLOCK_SIZE);

		cudaStatus = hipSetDevice(0);
		_checkCudaStatus("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");

		cudaStatus = hipMalloc((void**)&cudaInImageArray,sizeof(unsigned char)*inImageWidth*inImageHeight*3);
		_checkCudaStatus("hipMalloc failed!");

		cudaStatus = hipMalloc((void**)&cudaOutImageArray,sizeof(unsigned char) * outImageWidth * outImageHeight * 3);
		_checkCudaStatus("hipMalloc failed!");

		cudaStatus = hipMalloc((void**)&cudaFaceOffset,sizeof(int) * 6 * 2);
		_checkCudaStatus("hipMalloc failed!");

		cudaStatus = hipMalloc((void**)&cudaFace3DCoordsFactor,sizeof(int) * 6 * 3 * 2);
		_checkCudaStatus("hipMalloc failed!");

		cudaStatus = hipMalloc((void**)&cudaFace3DCoordsConstant,sizeof(int)*6*3);
		_checkCudaStatus("hipMalloc failed!");

		cudaStatus = hipMemcpy(cudaInImageArray,inImageArray,sizeof(unsigned char)*inImageWidth*inImageHeight*3,hipMemcpyHostToDevice);
		_checkCudaStatus("hipMemcpy failed!");

		cudaStatus = hipMemcpy(cudaOutImageArray,outImageArray,sizeof(unsigned char)*outImageWidth*outImageHeight*3,hipMemcpyHostToDevice);
		_checkCudaStatus("hipMemcpy failed!");

		cudaStatus = hipMemcpy(cudaFaceOffset,faceOffset,sizeof(int)*6*2,hipMemcpyHostToDevice);
		_checkCudaStatus("hipMemcpy failed!");

		cudaStatus = hipMemcpy(cudaFace3DCoordsFactor,face3DCoordsFactor,sizeof(int)*6*3*2,hipMemcpyHostToDevice);
		_checkCudaStatus("hipMemcpy failed!");

		cudaStatus = hipMemcpy(cudaFace3DCoordsConstant,face3DCoordsConstant,sizeof(int)*6*3,hipMemcpyHostToDevice);
		_checkCudaStatus("hipMemcpy failed!");

#if TIMER
		GpuTimer timer;
		timer.Start();
#endif

		pixelWarpKernelGPU<<<dimGrid, dimBlock>>>(cudaInImageArray,cudaOutImageArray,(int (*)[2])cudaFaceOffset,
			(int (*) [3][2])cudaFace3DCoordsFactor,
			(int (*) [3])cudaFace3DCoordsConstant,
			outBlockWidth);

#if TIMER
		timer.Stop();
		printf("GPU Warp kernel: %f ms.\n", timer.Elapsed());
#endif

		cudaStatus = hipDeviceSynchronize();
		_checkCudaStatus("hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);

		cudaStatus = hipMemcpy(outImageArray,cudaOutImageArray,sizeof(unsigned char)*outImageWidth*outImageHeight*3,hipMemcpyDeviceToHost);
		_checkCudaStatus("hipMemcpy failed!");

	Error:
		hipFree(cudaInImageArray);
		hipFree(cudaOutImageArray);
		hipFree(cudaFaceOffset);
		hipFree(cudaFace3DCoordsFactor);
		hipFree(cudaFace3DCoordsConstant);
		return cudaStatus;
	}

	int GPUmain(IplImage *inImageMat, IplImage **outImageMat) {
		hipError_t cudaStatus;

		int inImageWidth = inImageMat -> width;
		int inImageHeight = inImageMat -> height;

		if (inImageWidth != (inImageHeight << 1)) {
			fprintf(stderr, "This image is not a standard sphere panorama image (2:1) !");
			return 1;
		}

		int outBlockWidth = inImageWidth >> 2;

		int outImageWidth = outBlockWidth * 4;
		int outImageHeight = outBlockWidth * 3;

		unsigned char *inImageArray = (unsigned char *)(inImageMat->imageData);
		unsigned char *outImageArray = (unsigned char *)malloc(sizeof(unsigned char) * outImageWidth * outImageHeight * 3);

		memset(outImageArray, 0, outImageWidth * outImageHeight);

		cudaStatus = initCuda();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "initCuda failed!");
			return 1;
		}

		cudaStatus = convertWithCuda(outBlockWidth, inImageArray, outImageArray);

		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "convertWithCuda failed!");
			return 1;
		}

		*outImageMat = cvCreateImageHeader(cvSize(outImageWidth, outImageHeight), IPL_DEPTH_8U, 3);
		cvSetData(*outImageMat, outImageArray, outImageWidth * 3);

		cudaStatus = hipDeviceReset();
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipDeviceReset failed!");
			return 1;
		}

		return 0;
	}
}